#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#include<hip/hip_runtime.h>
#include<>

#define max_num 1000

__global__ void count(int* vector, int* result, int* class_vector,int n, int b)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x*gridDim.x;
	
	while(i < n)
	{
		atomicAdd(&result[vector[i]],1);

		atomicAdd(&class_vector[vector[i]/b],1);
		
		i += stride;
	}
	
}

int main(void)
{
	int i,b;
	int num_num;
	int num_class;
	int *vector_CPU;
	int *vector_GPU;
	int *result;
	int *final_result;
	int *class_vector;
	int *GPU_class_vector;
	hipError_t err;
	struct timeval start,end;
	final_result = (int*)malloc(max_num*sizeof(int));
	for(i = 0; i < max_num; i++)
	{
		final_result[i] = 0;
	}
	
	printf("How many numbers you want to use?\n");
	scanf("%d",&num_num);
	printf("How many class you will use?\n");
	scanf("%d",&num_class);
	vector_CPU = (int*)malloc(num_num*sizeof(int));
	class_vector = (int*)malloc(num_class*sizeof(int));
	hipMalloc((void**)&vector_GPU, num_num*sizeof(int));
	hipMalloc((void**)&result, max_num*sizeof(int));
	hipMalloc((void**)&GPU_class_vector, num_class*sizeof(int));
	srand((unsigned int)time(NULL));
	for(i = 0; i < num_num; i++) 	
	{
		vector_CPU[i] = rand()%max_num;
	}
	for(i = 0; i < num_class; i++)
	{
		class_vector[i] = 0;
	}

	gettimeofday(&start,NULL);
	
	err = hipMemcpy(vector_GPU, vector_CPU, num_num*sizeof(int), hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{printf("error1, code: %d, error: %s\n", err, hipGetErrorString(err));}
	
	err = hipMemcpy(GPU_class_vector, class_vector, num_class*sizeof(int), hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{printf("error1.5, code %d, error: %s\n", err, hipGetErrorString(err));}
	
	int threadsPerBlock = 256;
	int blocksPerGrid = (num_num + threadsPerBlock - 1)/threadsPerBlock;
	int share_size = (max_num+num_class)*sizeof(int);
	b = max_num / num_class;
	
	count<<<blocksPerGrid,threadsPerBlock,share_size>>>(vector_GPU, result, GPU_class_vector, num_num, b);
	err = hipGetLastError();
	if(err != hipSuccess)
	{printf("GPU failed to process!\n");}
		
	err = hipMemcpy(final_result, result, max_num*sizeof(int), hipMemcpyDeviceToHost);
	if(err != hipSuccess)
	{printf("error2, code: %d, error: %s\n", err, hipGetErrorString(err));}
	
	err = hipMemcpy(class_vector, GPU_class_vector, num_class*sizeof(int),hipMemcpyDeviceToHost);
	if(err != hipSuccess)
	{printf("error3, code: %d, error: %s\n", err, hipGetErrorString(err));}
	
	gettimeofday(&end,NULL);
	int k = 0;
	for(i = 0; i < max_num; i++)
	{
		if(final_result[i] != 0)
		{
			printf("Have %d number %d in class %d; ", final_result[i],i+1, (i+1)/b);
			k = k + 1;
			if(k == 2)
			{
				printf("\n");
				k = 0;
			}
		}
	}
	if(k < 4)
	{printf("\n");}
	k = 0;
	for(i = 0 ; i < num_class; i++)
	{
		printf("class %d have %d numbers; ", i+1, class_vector[i]);
		k = k + 1;
		if(k == 2);
		{printf("\n");
		k = 0;}
	}
	if(k!= 2)
	{printf("\n");}
	printf("Time is: %d us\n", end.tv_usec - start.tv_usec);
	hipFree(vector_GPU);
	hipFree(result);
	hipFree(GPU_class_vector);
	free(vector_CPU);
	free(final_result);
	free(class_vector);
	return 0;
}
