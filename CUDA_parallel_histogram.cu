#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#include<hip/hip_runtime.h>
#include<>

#define max_num 1000 //the maximum number the program can randomly generte

__global__ void count(int* vector, int* result, int* class_vector,int n, int b) //cuda kernal function
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x*gridDim.x;
	
	while(i < n)
	{
		atomicAdd(&result[vector[i]],1); //this is the atomic add, it is very important in histogram generating when we using cuda 

		atomicAdd(&class_vector[vector[i]/b],1);// result is used to save the number we have, class_vector is used to save how many number in each class
		
		i += stride;
	}
	
}

int main(void)
{
	int i,b;
	int num_num;
	int num_class;
	int *vector_CPU;
	int *vector_GPU;
	int *result;
	int *final_result;
	int *class_vector;
	int *GPU_class_vector;
	hipError_t err;
	struct timeval start,end;
	final_result = (int*)malloc(max_num*sizeof(int));
	for(i = 0; i < max_num; i++)
	{
		final_result[i] = 0;
	}
	
	printf("How many numbers you want to use?\n");
	scanf("%d",&num_num);
	printf("How many class you will use?\n");
	scanf("%d",&num_class);
	vector_CPU = (int*)malloc(num_num*sizeof(int)); //used to save the randomly generated value
	class_vector = (int*)malloc(num_class*sizeof(int)); //used to save how many numbers in eacch class
	hipMalloc((void**)&vector_GPU, num_num*sizeof(int)); // assign memory to the vector which will be used in GPU
	hipMalloc((void**)&result, max_num*sizeof(int));
	hipMalloc((void**)&GPU_class_vector, num_class*sizeof(int));
	srand((unsigned int)time(NULL)); // used to generate randm number
	for(i = 0; i < num_num; i++) 	
	{
		vector_CPU[i] = rand()%max_num; //generate random number
	}
	for(i = 0; i < num_class; i++)
	{
		class_vector[i] = 0; //initial vector
	}

	gettimeofday(&start,NULL); //save time
	
	err = hipMemcpy(vector_GPU, vector_CPU, num_num*sizeof(int), hipMemcpyHostToDevice);
	if(err != hipSuccess) //check whether our memory allocation is correct
	{printf("error1, code: %d, error: %s\n", err, hipGetErrorString(err));}
	
	err = hipMemcpy(GPU_class_vector, class_vector, num_class*sizeof(int), hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{printf("error1.5, code %d, error: %s\n", err, hipGetErrorString(err));}
	
	int threadsPerBlock = 256; //number of threads in each block
	int blocksPerGrid = (num_num + threadsPerBlock - 1)/threadsPerBlock; //number of blocks per grid
	int share_size = (max_num+num_class)*sizeof(int); //we don't need it, you can delete this sentence
	b = max_num / num_class; //this means how many numbers will assgn in each class
	
	count<<<blocksPerGrid,threadsPerBlock,share_size>>>(vector_GPU, result, GPU_class_vector, num_num, b);
	err = hipGetLastError();
	if(err != hipSuccess) //check whether the kernal function runs right or not
	{printf("GPU failed to process!\n");}
		
	err = hipMemcpy(final_result, result, max_num*sizeof(int), hipMemcpyDeviceToHost); //copy vector back
	if(err != hipSuccess)
	{printf("error2, code: %d, error: %s\n", err, hipGetErrorString(err));}
	
	err = hipMemcpy(class_vector, GPU_class_vector, num_class*sizeof(int),hipMemcpyDeviceToHost);
	if(err != hipSuccess)
	{printf("error3, code: %d, error: %s\n", err, hipGetErrorString(err));}
	
	gettimeofday(&end,NULL);
	int k = 0;
	//print all the information
	for(i = 0; i < max_num; i++)
	{
		if(final_result[i] != 0)
		{
			printf("Have %d number %d in class %d; ", final_result[i],i+1, (i+1)/b);
			k = k + 1;
			if(k == 2)
			{
				printf("\n");
				k = 0;
			}
		}
	}
	if(k < 4)
	{printf("\n");}
	k = 0;
	for(i = 0 ; i < num_class; i++)
	{
		printf("class %d have %d numbers; ", i+1, class_vector[i]);
		k = k + 1;
		if(k == 2);
		{printf("\n");
		k = 0;}
	}
	if(k!= 2)
	{printf("\n");}
	printf("Time is: %d us\n", end.tv_usec - start.tv_usec);
	//free the memory, never forget this step
	hipFree(vector_GPU);
	hipFree(result);
	hipFree(GPU_class_vector);
	free(vector_CPU);
	free(final_result);
	free(class_vector);
	return 0;
}
